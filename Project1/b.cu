#include "hip/hip_runtime.h"
#include <iostream>  
#include "hip/hip_runtime.h"  
#include ""  
using namespace std;
__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}
int main(){
	int c;
	int *dev_c;
	hipMalloc((void**)&dev_c, sizeof(int));
	add << <1, 1 >> >(5, 9, dev_c);
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	cout << "5 + 9 = " << c << endl;
	hipFree(dev_c);

	system("pause");
}