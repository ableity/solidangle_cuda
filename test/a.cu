#include "hip/hip_runtime.h"
/*
�����Ϸ���ʦ���matlab���������ϵͳ�������
v 1.0 
��ʹ��cuda��c�����
ע��matlab�����1��ʼ��c��0��ʼ
by ����
*/
#include <iostream>  
#include <string>
#include "hip/hip_runtime.h"  
#include ""  
//#pragma comment(lib,"cudart.lib")
#include "G:\cuda\bitmap\common\cpu_bitmap.h"
#include "G:\cuda\bitmap\common\book.h"
using namespace std;

void showarray(double *a,int n)
{
	if (n < 6)
	{
		printf("[");
		for (int i = 0; i < n; i++)
		{
			if (i != 0)
				printf(",");
			printf("%.4f", a[i]);
		}
		printf("]");
	}
	else
	{
		printf("[");
		for (int i = 0; i < 3; i++)
		{
			if (i != 0)
				printf(",");
			printf("%.4f", a[i]);
		}
		printf("...");
		for (int i = n-4; i < n; i++)
		{
			if (i != n-4)
				printf(",");
			printf("%.4f", a[i]);
		}
		printf("]\n");
	}
}

__global__ void initvalue(double *a, int n, double value)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int tid = blockDim.x * gridDim.x;
	while (i < n)
	{
		a[i] = value;
		i += tid;
	}
}


double* init_big_array(int num, double value)
{
	double *in;
	//�������ʼ��������cuda
	in = (double*)malloc(num*sizeof(double));
	double *in_cuda;
	HANDLE_ERROR(hipMalloc((void**)&in_cuda, num*sizeof(double)));
	initvalue << <1000, 1000 >> >(in_cuda, num, value);
	HANDLE_ERROR(hipMemcpy(in, in_cuda, num*sizeof(double), hipMemcpyDeviceToHost));
	return in;
}

double sum_array(double *a, int n, int m)
{
	double out = 0;
	for (int i = n; i <= m; i++)
	{
		out += a[i];
	}
	return out;
}

int main()
{
	//��������
	const int CryNumY = 77, CryNumZ = 104;
	//����ߴ�
	const int CrySize[3] = {26,4,4}; 
	//Y�ᾧ������
	double CryCoorY[CryNumY];
	for (int i = 0; i < CryNumY; i++)
	{
		//��ֵ��cuda ����֤
		CryCoorY[i] = -(double)CryNumY*(double)CrySize[1] / 2 + (double)CrySize[1]/2+(double)i*(double)CrySize[1];
	}
	double CryCoorZ[CryNumZ];
	for (int i = 0; i < CryNumZ; i++)
	{
		//��ֵ��cuda ����֤
		CryCoorZ[i] = -(double)CryNumZ*(double)CrySize[2] / 2 + (double)CrySize[2]/2+(double)i*(double)CrySize[2];
	}

	//showarray(CryCoorZ,104);
	//ÿ��̽�����ľ�����
	const int CryNumPerHead = CryNumY * CryNumZ;
	//����̽�����ľ���
	const double Dis = 240;
	//LOR����
	const int LORNum = CryNumY*CryNumY * CryNumZ*CryNumZ;
	const int VoxNumX = 240, VoxNumY = 308, VoxNumZ = 416;
	const int VoxNumYZ = VoxNumY * VoxNumZ;
	const int VoxSize = 1;
	double VoxCoorX[VoxNumX],VoxCoorY[VoxNumY],VoxCoorZ[VoxNumZ];
	for (int i = 0; i < VoxNumX; i++)
	{
		//cuda ����֤
		VoxCoorX[i] = -(double)VoxNumX*(double)VoxSize / 2 + (double)VoxSize / 2 + (double)i*(double)VoxSize;
	}
	for (int i = 0; i < VoxNumY; i++)
	{
		VoxCoorY[i] = -(double)VoxNumY*(double)VoxSize / 2 + (double)VoxSize / 2 + (double)i*(double)VoxSize;
	}
	for (int i = 0; i < VoxNumZ; i++)
	{
		VoxCoorZ[i] = -(double)VoxNumZ*(double)VoxSize / 2 + (double)VoxSize / 2 + (double)i*(double)VoxSize;
	}

	double gap = 0.22;
	const int VoxNum = VoxNumYZ*VoxNumX;

	double nonzero_ratio[13]={0.0823, 0.1036, 0.1015, 0.0971, 0.0914, 0.0854, 0.0794, 0.0736, 0.0680, 0.0627, 0.0575, 0.0522, 0.0453 };
	double theta = 1;


	//cuda ����֤��ֵ
	double DeltaWeight[4] = { nonzero_ratio[0], sum_array(nonzero_ratio, 1, 2), sum_array(nonzero_ratio, 4, 6), sum_array(nonzero_ratio, 7, 12) };
	double DeepLen[4] = { 0, 2, 6, 14 };

	double offAbandon = 0;
	double Start = 0;

	//���弰��ʼ��
	double *norm = init_big_array(LORNum, 0);

	double u_LYSO = 0.087;
	double coeff = 1;

	int LORi = 1;
	int LORj = 1;


	//MATLAB��1��ʼ��Ϊ�˳������еط����������ö���һ�������1��ʼ���еĵط����еĲ��������׼�飩
	for (int LORm = 1; LORm <= CryNumZ; LORm++)
	{
		for (int LORn = 1; LORn <= CryNumY; LORn++)
		{

		}
	}


	system("pause");
	return 0;

}


